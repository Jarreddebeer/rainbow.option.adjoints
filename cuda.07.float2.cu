#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iomanip>
#include <vector>
#include "asset.h"
#include "second.h"

#define BLOCKSIZE 128

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

static __global__ void adjoint_method_correlation_GPU(
    float2* d_ST_max, 
    float2* d_ST_del,
    float2* d_ST_veg, 
    int2* d_ST_aid,
    float* d_assets,
    float* d_chlsky,
    int num_sims,
    int num_steps,
    int num_assets,
    float dt,
    float r,
    float K
) {

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float2 shared_mem[12*BLOCKSIZE];
    // assets:0->17, chlsky:18->26, Z_indp:28->(28+6*BLOCKSIZE), x23:(28+6*BLOCKSIZE)->(28+9*BLOCKSIZE), x24:(28+9*BLOCKSIZE)->(28+12*BLOCKSIZE)
    // note: Z_indp must be aligned in memory, this is why index '27' is skipped when assigning Z_indp


if (tid < num_sims) {

//    float* __restrict__ assets = &shared_mem[0]; // 3 * 6 [0-17]
//    float* __restrict__ chlsky = &shared_mem[18]; // 3 * 3 [18-26];
    float2* __restrict__ Z_indp = &shared_mem[6*threadIdx.x];
    float2* __restrict__ x23    = &shared_mem[6*BLOCKSIZE+3*threadIdx.x];
    float2* __restrict__ x24    = &shared_mem[9*BLOCKSIZE+3*threadIdx.x];

    //float2 Z_indp[3];
    float2  Z_corr[3];
    float assets[18];
    float chlsky[9];

    int2 winning_asset;
    float2 ST_max, ST_del, ST_veg;

    float2 x_8, x_7, x_6, x_5, x_4, x_3, x_2, x_1;
    float2 x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15, x16, x17, x18, x19, x20, x21, x22;
    float2 _x24, _x22, _x19, _x15, _x9, _x0;
    float2 _x23, _x21, _x17, _x16, _x13, _x11, _x8, _x5v, _x0v;
    float2 _x_1[3], _x_2[3], _x_2v[3];
    float2 Zs, Zv;
    float2 payoff;

    // fetch asset and chlsky data
    for (int i=0; i<18; i++) { assets[i] = d_assets[i]; }
    for (int i=0; i<9; i++)  { chlsky[i] = d_chlsky[i]; }
//    if (threadIdx.x < 18) { shared_mem[threadIdx.x] = d_assets[threadIdx.x]; }
//    if (threadIdx.x < 9)  { shared_mem[18+threadIdx.x] = d_chlsky[threadIdx.x]; } 
    // done fetching
     
    // random number generator and state
    hiprandStatePhilox4_32_10 rng_state;
    hiprand_init(1234, tid, 0, &rng_state);

    winning_asset = make_int2(0, 0);
    ST_max = make_float2(0.0f, 0.0f);
    ST_del = make_float2(0.0f, 0.0f);
    ST_veg = make_float2(0.0f, 0.0f);

    for (int a=0; a<num_assets; a++) {

        x23[a] = make_float2(assets[1*num_assets+a],
                             assets[1*num_assets+a]); // vega  

        x24[a] = make_float2(assets[0*num_assets+a],
                             assets[0*num_assets+a]); // asset

        // init sensitivity vector
        _x_1[a] = make_float2(1.0f, 1.0f);
        _x_2[a] = make_float2(0.0f, 0.0f);
       _x_2v[a] = make_float2(1.0f, 1.0f); 

     }

     for (int t=0; t<num_steps; t++)
     {
           // generate correlated rngs
         for (int a=0; a<num_assets; a++)
             { reinterpret_cast<float4*>(&Z_indp[4*a])[0] = hiprand_normal4(&rng_state); Z_corr[a] = make_float2(0.0f, 0.0f); }
         for (int row=0; row<num_assets; row++) for (int c=0; c<num_assets; c++)
             { Z_corr[row] = make_float2(Z_corr[row].x + chlsky[num_assets*row+c] * Z_indp[2*c].x,
                                         Z_corr[row].y + chlsky[num_assets*row+c] * Z_indp[2*c].y); }

         for (int a=0; a<num_assets; a++)
         {
          
            // init input parameters
            x_8 = make_float2(assets[2*num_assets+a], assets[2*num_assets+a]); // rho
            x_7 = make_float2(assets[3*num_assets+a], assets[3*num_assets+a]); // kappa
            x_6 = make_float2(assets[4*num_assets+a], assets[4*num_assets+a]); // theta
            x_5 = make_float2(assets[5*num_assets+a], assets[5*num_assets+a]); // sigma
            x_4 = make_float2(dt, dt);
            x_3 = make_float2(r, r);
            x_2 = x23[a];
            x_1 = x24[a];

            Zs = Z_corr[a];
            Zv = make_float2(x_8.x * Zs.x + sqrt(1-x_8.x*x_8.x) * Z_indp[2*a+1].x,
                             x_8.y * Zs.y + sqrt(1-x_8.y*x_8.y) * Z_indp[2*a+1].y);

            // forward pass

            x0 = make_float2(x_2.x * x_4.x,
                             x_2.y * x_4.y);

            x1 = make_float2(x_3.x * x_4.x,
                             x_3.y * x_4.y);

            x2 = make_float2(x_6.x * x_4.x,
                             x_6.y * x_4.y);

            x3 = make_float2(x_5.x * x_5.x,
                             x_5.y * x_5.y);

            x4 = make_float2(Zv.x * Zv.x,
                             Zv.y * Zv.y);

            x5 = make_float2(sqrt(x0.x),
                             sqrt(x0.y));

            x6 = make_float2(x3.x * x_4.x,
                             x3.y * x_4.y);

            x7 = make_float2(x_7.x * x2.x,
                             x_7.y * x2.y);

            x8 = make_float2(x_7.x * x0.x,
                             x_7.y * x0.y);

            x9 = make_float2(-0.5f * x0.x,
                             -0.5f * x0.y);

            x10 = make_float2(x4.x - 1,
                              x4.y - 1);

            x11 = make_float2(x_5.x * x5.x,
                              x_5.y * x5.y);

            x12 = make_float2(Zs.x * x5.x,
                              Zs.y * x5.y);

            x13 = make_float2(x_2.x + x7.x,
                              x_2.y + x7.y);

            x14 = make_float2(0.25f * x10.x,
                              0.25f * x10.y);

            x15 = make_float2(x9.x + x12.x,
                              x9.y + x12.y);

            x16 = make_float2(Zv.x * x11.x,
                              Zv.y * x11.y);

            x17 = make_float2(x13.x - x8.x,
                              x13.y - x8.y);

            x18 = make_float2(x14.x * x6.x,
                              x14.y * x6.y);

            x19 = make_float2(x1.x + x15.x,
                              x1.y + x15.y);

            x20 = make_float2(x17.x + x16.x,
                              x17.y + x16.y);

            x21 = make_float2(x20.x + x18.x,
                              x20.y + x18.y);

            x22 = make_float2(exp(x19.x),
                              exp(x19.y));

            x23[a] = make_float2(max(x21.x, 0.0f),
                                 max(x21.y, 0.0f));

            x24[a] = make_float2(x_1.x * x22.x,
                                 x_1.y * x22.y);


            // adjoint pass

            // asset adjoints

            _x24 = make_float2(1, 1);
            _x22 = make_float2(x_1.x * _x24.x,
                               x_1.y * _x24.y);

            _x19 = make_float2(x22.x * _x22.x,
                               x22.y * _x22.y);

            _x15 = _x19;
            _x9 = _x15;
            _x0 = make_float2(-0.5f * _x9.x,
                              -0.5f * _x9.y);


            // volatility adjoints
            _x23 = make_float2(1.0f, 1.0f);
            _x21 = make_float2((x23[a].x > 0.0f) ? _x23.x : 0.0f,
                               (x23[a].y > 0.0f) ? _x23.y : 0.0f);

            _x13 = _x16 = _x17 = _x21;

            _x11 = make_float2(Zv.x * _x16.x,
                               Zv.y * _x16.y);

            _x8 = make_float2(-1 * _x17.x,
                              -1 * _x17.y);

            _x5v = make_float2(x_5.x * _x11.x,
                               x_5.y * _x11.y);

            _x0v = make_float2((0.5f * _x5v.x / x5.x) + _x8.x * x_7.x,
                               (0.5f * _x5v.y / x5.y) + _x8.y * x_7.y);


            // xbar inputs
            _x_1[a] = make_float2((_x24.x * x22.x) * _x_1[a].x,
                                  (_x24.y * x22.y) * _x_1[a].y);

            _x_2[a] = make_float2(_x_2[a].x * x22.x - (_x0.x *x_4.x) * _x_2v[a].x * (-1 + Zs.x / x5.x),
                                  _x_2[a].y * x22.y - (_x0.y *x_4.y) * _x_2v[a].y * (-1 + Zs.y / x5.y));

            _x_2v[a] = make_float2(_x_2v[a].x * (_x13.x + _x0v.x * x_4.x),
                                   _x_2v[a].y * (_x13.y + _x0v.y * x_4.y));


         }
     }

     for (int a=0; a<num_assets; a++) {

         payoff = make_float2(max(x24[a].x-K, 0.0f),
                              max(x24[a].y-K, 0.0f));

         if (payoff.x > ST_max.x) {
             winning_asset.x = a;
             ST_max.x = payoff.x;
             ST_del.x = _x_1[a].x;
             ST_veg.x = _x_2[a].x;
         }

         if (payoff.y > ST_max.y) {
             winning_asset.y = a;
             ST_max.y = payoff.y;
             ST_del.y = _x_1[a].y;
             ST_veg.y = _x_2[a].y;
         }

     }

     d_ST_max[tid] = ST_max;
     d_ST_del[tid] = ST_del;
     d_ST_veg[tid] = ST_veg;
     d_ST_aid[tid] = winning_asset;

} // end if (tid < num_sims)
}


int main(int argc, char **argv)
{

    int num_sims, num_steps, num_assets;
    float dt, r, K, T;
    double overhead, start, duration, price, delta[3], vega[3];

    num_sims  = strtod(argv[1], NULL);
    num_steps = strtod(argv[2], NULL);

    num_sims /= 2;

    cin >> num_assets;

    dim3 dimBlock(BLOCKSIZE, 1, 1);
    dim3 dimGrid(ceil( ((float)num_sims)/BLOCKSIZE ), 1, 1);
    int ST_bytes = num_sims * sizeof(float2);
    int aid_bytes = num_sims * sizeof(int2);
    int assets_bytes = num_assets * 6 * sizeof(float);
    int chlsky_bytes = num_assets * num_assets * sizeof(float);

    // allocate memory for gpu and host
    float2* h_ST_max = (float2*) malloc(ST_bytes);
    float2* h_ST_del = (float2*) malloc(ST_bytes);
    float2* h_ST_veg = (float2*) malloc(ST_bytes);
    int2*    h_ST_aid = (int2*)    malloc(aid_bytes);
    float2* d_ST_max = (float2*) malloc(ST_bytes);
    float2* d_ST_del = (float2*) malloc(ST_bytes);
    float2* d_ST_veg = (float2*) malloc(ST_bytes);
    int2*    d_ST_aid = (int2*)    malloc(aid_bytes);
    //
    float* h_assets = (float*) malloc(assets_bytes);
    float* h_chlsky = (float*) malloc(chlsky_bytes);
    float* d_assets = (float*) malloc(assets_bytes);
    float* d_chlsky = (float*) malloc(chlsky_bytes);
    //
    gpuErrchk( hipMalloc((void**) &d_ST_max, ST_bytes) );
    gpuErrchk( hipMalloc((void**) &d_ST_del, ST_bytes) );
    gpuErrchk( hipMalloc((void**) &d_ST_veg, ST_bytes) );
    gpuErrchk( hipMalloc((void**) &d_ST_aid, aid_bytes) );
    gpuErrchk( hipMalloc((void**) &d_assets, assets_bytes) );
    gpuErrchk( hipMalloc((void**) &d_chlsky, chlsky_bytes) );

    // read asset parameters
    for (int i=0; i<num_assets; i++) {
        asset a;
        cin >> a.S >> a.V >> a.r >> a.T >> a.kappa >> a.theta >> a.sigma >> a.rho >> a.K;
        // these two are constant between assets (i'm being lazy here by re-assigning)
        T = a.T;
        dt = T / num_steps;
        r = a.r;
        K = a.K;
        //
        h_assets[0*num_assets+i] = a.S;
        h_assets[1*num_assets+i] = a.V;
        h_assets[2*num_assets+i] = a.rho;
        h_assets[3*num_assets+i] = a.kappa;
        h_assets[4*num_assets+i] = a.theta;
        h_assets[5*num_assets+i] = a.sigma;
    }

    // read lower cholesky decomposed matrix
    for (int i=0; i<num_assets; i++) {
        for (int j=0; j<num_assets; j++) {
            cin >> h_chlsky[num_assets*i+j];
        }
    }

    overhead = second()-second(); // overhead of timing method
    start = second();

    // copy data across
    gpuErrchk( hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte) );
    gpuErrchk( hipMemcpy(d_assets, h_assets, assets_bytes, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_chlsky, h_chlsky, chlsky_bytes, hipMemcpyHostToDevice) );

    adjoint_method_correlation_GPU<<<dimGrid, dimBlock>>>(
        d_ST_max, d_ST_del, d_ST_veg, d_ST_aid, d_assets, d_chlsky, num_sims, num_steps, num_assets, dt, r, K
    );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipMemcpy(h_ST_max, d_ST_max, ST_bytes, hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_ST_del, d_ST_del, ST_bytes, hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_ST_veg, d_ST_veg, ST_bytes, hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_ST_aid, d_ST_aid, aid_bytes, hipMemcpyDeviceToHost) );


    for (int i=0; i<num_sims; i++) {
        price += (h_ST_max[i].x + h_ST_max[i].y);
        delta[(int)h_ST_aid[i].x] += h_ST_del[i].x;
        delta[(int)h_ST_aid[i].y] += h_ST_del[i].y;
         vega[(int)h_ST_aid[i].x] += h_ST_veg[i].x;
         vega[(int)h_ST_aid[i].y] += h_ST_veg[i].y;
    }    
    double disc_fac = exp(-r*T);
    num_sims *= 2;
    price = disc_fac * price / num_sims;

    printf("--------------------------------------\n");
    printf("Heston 3 assets rainbow call on max\n");
    printf("price: %0.15g\n", price);
    for (int i=0; i<num_assets; i++) {
        printf("delta %d: %0.15g\n", disc_fac * delta[i] / num_sims);
        printf("vega %d: %0.15g\n",  disc_fac * vega[i]  / num_sims);
    }

    duration = second()-start-overhead;

    printf("======================================\n");
    printf("duration: %0.15g\n", duration);
    printf("======================================\n");

    return 0;
}
